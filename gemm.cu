	//header files included
	#include <fstream>
	#include <iostream>
	#include <stdio.h>
	#include <string>
	#include <sstream>
	#include <stdlib.h>
	#include <math.h>
	#include <time.h>
	#include <ctime>
	#include <vector>
	#include <hip/hip_runtime.h>
	#include <hip/hip_runtime_api.h>
	#include <hipblas.h>

	//declaring the tile width and height 
	//for tile based matrix multiplication
	#define TILE_WIDTH 32
	#define TILE_HEIGHT 32
	
	//Namespace for std
	using namespace std;

	//structure declaration for storing rows and columns for a matrix
	struct matrix{
		unsigned int rows;	//storing rows of a matrix
		unsigned int cols;	//storing columns of a matrix
	};

	//handlerror declaration : to display file and line numbers of erroneous lines
	static void HandleError( hipError_t err, const char *file, int line ) {
		if (err != hipSuccess) {
			cout<<hipGetErrorString(err)<<" in "<< file <<" at line "<< line<<endl;
		}
	}

	//handle error alias name declaration
	#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

	//global kernal for matrix multiplication, takes in input matrices and sizes, and multiplies them
	//matrix multiplication is being done tile by tile
	__global__ void matrix_mult(float* array1, unsigned int rows1, unsigned int cols1, float* array2, unsigned int rows2, unsigned int cols2, float* array3)
	{	
		//shared memory takes one tile at a time
		__shared__ float S1[TILE_WIDTH][TILE_HEIGHT];	//to store tiles for array 1
		__shared__ float S2[TILE_HEIGHT][TILE_WIDTH];	//to store tiles for array 2

		//threads x and y index for the current block
		unsigned int tx=threadIdx.x;	
		unsigned int ty=threadIdx.y;

		unsigned int c=blockIdx.x*blockDim.x + threadIdx.x;	//row value using x-index of current thread
		unsigned int r=blockIdx.y*blockDim.y + threadIdx.y;	//column value using y-index of current thread

		unsigned int idx=c*rows1+r;				//column major index, using row and column value
		
		float val=0;		//register to store multiplication result initialized to zero

		for(int m=0; m<1+((rows2-1)/TILE_WIDTH);m++)	//going over all tiles one by one, with each m
		{

			int var1=m*TILE_WIDTH+tx ;		//x thread value for current tile
			int var2=m*TILE_WIDTH+ty ;		//y thread value for current tile
			
			//copying a tile from array1
			if (r < rows1 && var1 < rows2)		//if the value is associated to a valid matrix coordinate in array1 then store it to shared memory S1
				S1[ty][tx]=array1[r + var1*rows1];//storing a "valid" value from array to shared memory
			else
					S1[ty][tx]=0;					//storing zero, since there is no valid value
       			__syncthreads();						//syncing all threads once shared memory S1 is stored
			
			//copying a tile from array2
	       		if(c < cols2 && var2 < rows2)	//if value is associates to a valid matrix coordinate in array2 then store it to shared memory S2
	      			S2[ty][tx]=array2[var2+rows2*c];	//storing the valid value
	      		else 
	      			S2[ty][tx]=0;		//storing zero, since no valid value
			__syncthreads();		//synchronizing threads
			

			for(int i=0; i<TILE_WIDTH;i++)	//going over entire tile, ty row in S1 and tx column in S2
				val+=S1[ty][i]*S2[i][tx];	//and multiplying elements
			__syncthreads();		//synchronizing threads

		}
		
		if(r < rows1 && c< cols2)	//removing degenerate cases
			array3[idx]=val;	//saving multiplication result to global memory
			
	}

	int main(int argc, char* argv[])
	{
		if(argc != 4) //there should be four arguments, Usage: prog matrix1.mtx matrix2.mtx matrix3.mtx
		return 1; //exit and return an error

		ifstream infile_A, infile_B;	//reading the input matrices
		
		//READING matrix A
		infile_A.open(argv[1],ios::binary|ios::in|ios::ate);
		
		//getting end and beginning of the file
		infile_A.seekg(0,ios::end);
		infile_A.seekg(0,ios::beg);
		
		//memory allocation
		matrix M_A;	//to store rows and cols
		infile_A.read(reinterpret_cast<char*>(&M_A),2*sizeof(unsigned int));//read cols and rows value
		

		float* array_A=(float*)malloc(M_A.rows*M_A.cols*sizeof(float));	//array_A stores the first matrix
		infile_A.read(reinterpret_cast<char*>(array_A),M_A.rows*M_A.cols*sizeof(float));//reading the matrix elements in column major format
		
		infile_A.close();//closing the first file

		//READING matrix B
		infile_B.open(argv[2],ios::binary|ios::in|ios::ate);
		
		//getting end and beginning of the file
		infile_B.seekg(0,ios::end);
		infile_B.seekg(0,ios::beg);
		
		//memory allocation
		matrix M_B;	//to store rows and cols for second matrix
		infile_B.read(reinterpret_cast<char*>(&M_B),2*sizeof(unsigned int));//reading cols and rows valus

		float* array_B=(float*)malloc(M_B.rows*M_B.cols*sizeof(float));	//array_B stores the second matrix

		infile_B.read(reinterpret_cast<char*>(array_B),M_B.rows*M_B.cols*sizeof(float));//reading the matrix elements in column major format
		
		infile_B.close();//closing second file after reading

		if(M_A.cols!=M_B.rows) //checking if the two matrices can be multiplied
		{
			cout<<"Illegal matrix sizes: "<<M_A.cols<<" != "<<M_B.rows<<endl;
			return 1;
		}

		float* array_C=(float*)malloc(M_A.rows*M_B.cols*sizeof(float));//array to store gpu result in column major format
		
		float* array_D=(float*)malloc(M_A.rows*M_B.cols*sizeof(float));//arary to store cublas result in column major format
		


		//GPU DEVICE PROPERTIES and selecting a GPU for calculation
		int nDevices;
		HANDLE_ERROR(hipGetDeviceCount(&nDevices));

		hipDeviceProp_t prop;
	   	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));	//using GPU0

	   	//BLOCK AND GRID SIZE DECLARATION
	   	float thread_block=sqrt(prop.maxThreadsPerBlock);	//2D blocks used
		dim3 DimGrid(ceil(M_B.cols/thread_block),ceil(M_A.rows/thread_block),1); //image saved as a 2D grid
		dim3 DimBlock(thread_block,thread_block,1);

		size_t Sbytes = 2* DimBlock.x * DimBlock.y ;	//2 arrays used in the calculation, hence 2 * DimBlock.x * DimBlock.y
		
		//Checking if sufficient shared memory available or not

		if(prop.sharedMemPerBlock < Sbytes){
			std::cout<<"ERROR: insufficient shared memory"<<std::endl;
			exit(1);
		}

		//GPU MEMORY ALLOCATION
		float *array_A_gpu, *array_B_gpu, *array_C_gpu, *array_D_gpu;	//gpu arrays declared
	   
		HANDLE_ERROR(hipMalloc(&array_A_gpu,M_A.rows*M_A.cols*sizeof(float))); //allocate space to store arrayA

		HANDLE_ERROR(hipMalloc(&array_B_gpu,M_B.rows*M_B.cols*sizeof(float))); //allocate space to store arrayB

		HANDLE_ERROR(hipMalloc(&array_C_gpu,M_A.rows*M_B.cols*sizeof(float))); //allocate space to store gpu result

		HANDLE_ERROR(hipMalloc(&array_D_gpu,M_A.rows*M_B.cols*sizeof(float))); //allocate space to store cublas result


		//COPY TO GPU MEMORY
		HANDLE_ERROR(hipMemcpy(array_A_gpu, array_A, M_A.rows*M_A.cols*sizeof(float), hipMemcpyHostToDevice));//copy arrayA to gpu

		HANDLE_ERROR(hipMemcpy(array_B_gpu, array_B, M_B.rows*M_B.cols*sizeof(float), hipMemcpyHostToDevice));//copy arrayB to gpu

		HANDLE_ERROR(hipMemcpy(array_C_gpu, array_C, M_A.rows*M_B.cols*sizeof(float), hipMemcpyHostToDevice));//copy arrayC to gpu

		HANDLE_ERROR(hipMemcpy(array_D_gpu, array_D, M_A.rows*M_B.cols*sizeof(float), hipMemcpyHostToDevice));//copy arrayD to gpu


		//time measurement for matrix multiplication
		hipEvent_t start1, stop1;
	 	
	 	hipEventCreate(&start1);
		hipEventCreate(&stop1);
		
		//MATRIX MULTIPLICATION USING KERNEL
		hipEventRecord(start1);
		matrix_mult<<<DimGrid, DimBlock, Sbytes>>>(array_A_gpu,M_A.rows,M_A.cols,array_B_gpu,M_B.rows,M_B.cols,array_C_gpu);//calling the kernel
		hipEventRecord(stop1);

		hipEventSynchronize(stop1);

		float milliseconds1 = 0, milliseconds2 = 0;//storing the execution time in milliseconds
		
		hipEventElapsedTime(&milliseconds1, start1, stop1);//get the time in milliseconds
		cout<<"time taken by GPU = "<<milliseconds1<<" ms"<<endl;//printing time taken by GPU

		//copy to CPU MEMORY
		HANDLE_ERROR(hipMemcpy(array_C, array_C_gpu, M_A.rows*M_B.cols*sizeof(float), hipMemcpyDeviceToHost));//copying result of multiplication from gpu to cpu

		//Creating handle for CUBLAS
		hipblasHandle_t handle;
		hipblasCreate(&handle);	

		//parameter declaration for cublas implementation
		float alpha = 1.0;
		float beta = 0.0;
	    
		//cublas time measurement
	    	hipEvent_t start2, stop2;
	 	
	 	hipEventCreate(&start2);
		hipEventCreate(&stop2);

		//MATRIX MULTIPLICATION USING CUBLAS 
		hipEventRecord(start2);
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M_A.rows, M_B.cols, M_A.cols, &alpha, array_A_gpu, M_A.rows, array_B_gpu, M_B.rows, &beta, array_D_gpu, M_A.rows);
		hipEventRecord(stop2);

		hipEventSynchronize(stop2);

		hipEventElapsedTime(&milliseconds2, start2, stop2);//get the time in milliseconds
		cout<<"time taken by CUBLAS= "<<milliseconds2<<" ms"<<endl;//printing time taken by CUBLAS
		
		//copy to CPU MEMORY
	    
	    	HANDLE_ERROR(hipMemcpy(array_D, array_D_gpu, M_A.rows*M_B.cols*sizeof(float), hipMemcpyDeviceToHost));//copy result of multiplication using CUBLAS from gpu to cpu

		//CALCULATING MEAN SQUARED ERROR IN BOTH METHODS OF MATRIX MULTIPLICATION
		float mse=0; //mean squared error;

		for(int i=0; i<M_A.rows*M_B.cols;i++)
			mse=mse+(array_C[i]-array_D[i])*(array_C[i]-array_D[i]);//calculating element by element
		
		mse=mse/(M_A.rows*M_B.cols);	//taking the mean of squared error
			
		cout<<endl<<"Mean square error = "<<mse<<endl;//printing out the mean squared error

		//SAVING THE OUTPUT MATRIX
		ofstream ofile(argv[3], ios::binary);

		ofile.write((char*) &M_A.rows, sizeof(unsigned int));//writing the rows
		ofile.write((char*) &M_B.cols, sizeof(unsigned int));//writing the cols
		ofile.write((char*) array_C , M_A.rows*M_B.cols*sizeof(float));//writing all elements


		return 0;
	}
