#include <fstream>
#include <iostream>
#include <stdio.h>
#include <string>
#include <sstream>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <ctime>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

using namespace std;

struct matrix{
	unsigned int rows;
	unsigned int cols;
};

static void HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		cout<<hipGetErrorString(err)<<" in "<< file <<" at line "<< line<<endl;
	}
}


#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__ void matrix_mult(float* array1, unsigned int rows1, unsigned int cols1, float* array2, unsigned int rows2, unsigned int cols2, float* array3)
{
	//float* array3=(float*)malloc(rows1*cols2*sizeof(float));
	
	size_t c=blockIdx.x*blockDim.x + threadIdx.x;
	size_t r=blockIdx.y*blockDim.y + threadIdx.y;

	size_t idx=r*rows1+c;

	//initailize the array to zero
	if(idx<rows1*cols2)
	{
		array3[idx]=0;
		for(int k=0;k<rows2;k++)
		{
			array3[idx]+=array1[rows1*k+r]*array2[rows2*c+k];
		}
	}	
	
	//return C;

}

int main(int argc, char* argv[])
{
	if(argc != 4) //there should be four arguments
	return 1; //exit and return an error

	time_t reading_start=time(NULL);

	ifstream infile_A, infile_B;	//reading the input matrices
	
	
	//READING matrix A
	infile_A.open(argv[1],ios::binary|ios::in|ios::ate);
	
	//getting end and beginning of the file
	infile_A.seekg(0,ios::end);
	infile_A.seekg(0,ios::beg);
	
	//memory allocation
	matrix M_A;
	infile_A.read(reinterpret_cast<char*>(&M_A),2*sizeof(unsigned int));
	//cout<<M_A.rows<<M_A.cols;
	
	float* array_A=(float*)malloc(M_A.rows*M_A.cols*sizeof(float));	//column major
	infile_A.read(reinterpret_cast<char*>(array_A),M_A.rows*M_A.cols);
	
	infile_A.close();

	//READING matrix B
	infile_B.open(argv[1],ios::binary|ios::in|ios::ate);
	
	//getting end and beginning of the file
	infile_B.seekg(0,ios::end);
	infile_B.seekg(0,ios::beg);
	
	//memory allocation
	matrix M_B;
	infile_B.read(reinterpret_cast<char*>(&M_B),2*sizeof(unsigned int));

	float* array_B=(float*)malloc(M_B.rows*M_B.cols*sizeof(float));	//column major
	infile_B.read(reinterpret_cast<char*>(array_B),M_B.rows*M_B.cols);
	
	infile_B.close();

	float* array_C=(float*)malloc(M_A.rows*M_B.cols*sizeof(float));//gpu result
	
	float* array_D=(float*)malloc(M_A.rows*M_B.cols*sizeof(float));//cublas result

	
	time_t reading_end = time(NULL);


	//GPU DEVICE PROPERTIES
	int nDevices;
	HANDLE_ERROR(hipGetDeviceCount(&nDevices));

	hipDeviceProp_t prop;
   	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));	//using GPU0

   	//BLOCK AND GRID SIZE
        float thread_block=sqrt(prop.maxThreadsPerBlock);
	dim3 DimGrid(ceil(M_B.cols/thread_block),ceil(M_A.rows/thread_block),1); //image saved as a 2D grid
	dim3 DimBlock(thread_block,thread_block,1);

	//GPU MEMORY ALLOCATION
	float *array_A_gpu, *array_B_gpu, *array_C_gpu, *array_D_gpu;
   
	HANDLE_ERROR(hipMalloc(&array_A_gpu,M_A.rows*M_A.cols*sizeof(float))); //allocate space to store convolution result

	HANDLE_ERROR(hipMalloc(&array_B_gpu,M_B.rows*M_B.cols*sizeof(float))); //allocate space to store convolution temporary

	HANDLE_ERROR(hipMalloc(&array_C_gpu,M_A.rows*M_B.cols*sizeof(float))); //allocate space to copy image to GPU memory

	HANDLE_ERROR(hipMalloc(&array_D_gpu,M_A.rows*M_B.cols*sizeof(float))); //allocate space to copy image to GPU memory

	

	//COPY TO GPU MEMORY
	HANDLE_ERROR(hipMemcpy(array_A_gpu, array_A, M_A.rows*M_A.cols*sizeof(float), hipMemcpyHostToDevice));//copy input image from global to gpu

	HANDLE_ERROR(hipMemcpy(array_B_gpu, array_B, M_B.rows*M_B.cols*sizeof(float), hipMemcpyHostToDevice));//copy the kernel0 host to device

	HANDLE_ERROR(hipMemcpy(array_C_gpu, array_C, M_A.rows*M_B.cols*sizeof(float), hipMemcpyHostToDevice));//copy kernel1 host to device

	HANDLE_ERROR(hipMemcpy(array_D_gpu, array_D, M_A.rows*M_B.cols*sizeof(float), hipMemcpyHostToDevice));//copy kernel1 host to device

	time_t memory_transfers=time(NULL);

	//MATRIX MULTIPLICATION

	matrix_mult<<<DimGrid,DimBlock>>>(array_A_gpu,M_A.rows,M_A.cols,array_B_gpu,M_B.rows,M_B.cols,array_C_gpu);


	time_t mult_end = time(NULL);


	//copy to CPU MEMORY
	HANDLE_ERROR(hipMemcpy(array_C, array_C_gpu, M_A.rows*M_B.cols*sizeof(float), hipMemcpyDeviceToHost));//copy kernel1 host to device

	//Creating handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);	

	float alpha = 1.0;
	float beta = 1.0;

	hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M_A.rows, M_B.cols, M_A.cols, &alpha, array_A_gpu, M_A.rows, array_B_gpu, M_B.rows, &beta, array_D_gpu, M_A.rows);

	//copy to CPU MEMORY
        HANDLE_ERROR(hipMemcpy(array_D, array_D_gpu, M_A.rows*M_B.cols*sizeof(float), hipMemcpyDeviceToHost));//copy kernel1 host to device

	float mse=0; //mean squared error

	for(int i=0; i<M_A.rows*M_B.cols;i++)
	{
		mse=mse+(array_C[i]-array_D[i])*(array_C[i]-array_D[i]);
		int diff=array_C[i]-array_D[i];
		cout<<diff<<" "<<array_C[i]<<" "<<array_D[i]<<"               ";
	}

	//for(int i=0;i<M_A.rows*M_A.cols;i++)
	//cout<<array_A[i]<<" ";

	cout<<"Mean square error = "<<mse<<endl;

	//SAVING THE OUTPUT MATRIX
	ofstream ofile(argv[3], ios::binary);

	ofile.write((char*) &M_A.rows, sizeof(unsigned int));
	ofile.write((char*) &M_B.cols, sizeof(unsigned int));	
	ofile.write((char*) array_C , M_A.rows*M_B.cols*sizeof(float))	;

	time_t saved = time(NULL);

	cout<<"Matrix reading     :"<<double(reading_end - reading_start)<<" secs"<<endl;
	cout<<"Memory Transfers   :"<<double(memory_transfers - reading_end)<<" secs"<<endl;
	cout<<"Multiplication done:"<<double(mult_end - memory_transfers)<<" secs"<<endl;
	cout<<"Matrix saving      :"<<double(saved - mult_end)<<" secs"<<endl;

	return 0;
}