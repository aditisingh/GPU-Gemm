	//header files included
	#include <fstream>
	#include <iostream>
	#include <stdio.h>
	#include <string>
	#include <sstream>
	#include <stdlib.h>
	#include <math.h>
	#include <time.h>
	#include <ctime>
	#include <vector>
	#include <hip/hip_runtime.h>
	#include <hip/hip_runtime_api.h>
	#include <hipblas.h>

	//declaring the tile width and height 
	//for tile based matrix multiplication
	#define TILE_WIDTH 32
	#define TILE_HEIGHT 32
	
	//Namespace for std
	using namespace std;

	//structure declaration for storing rows and columns for a matrix
	struct matrix{
		unsigned int rows;	//storing rows of a matrix
		unsigned int cols;	//storing columns of a matrix
	};

	//handlerror declaration : to display file and line numbers of erroneous lines
	static void HandleError( hipError_t err, const char *file, int line ) {
		if (err != hipSuccess) {
			cout<<hipGetErrorString(err)<<" in "<< file <<" at line "<< line<<endl;
		}
	}

	//handle error alias name declaration
	#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

	//global kernal for matrix multiplication, takes in input matrices and sizes, and multiplies them
	//matrix multiplication is being done tile by tile
	__global__ void matrix_mult(float* array1, unsigned int rows1, unsigned int cols1, float* array2, unsigned int rows2, unsigned int cols2, float* array3)
	{	
		//shared memory takes one tile at a time
		__shared__ float S1[TILE_WIDTH][TILE_HEIGHT];	//to store tiles for array 1
		__shared__ float S2[TILE_HEIGHT][TILE_WIDTH];	//to store tiles for array 2

		//threads x and y index for the current block
		unsigned int tx=threadIdx.x;
		unsigned int ty=threadIdx.y;

		unsigned int c=blockIdx.x*blockDim.x + threadIdx.x;	//row value using x-index of current thread
		unsigned int r=blockIdx.y*blockDim.y + threadIdx.y;	//column value using y-index of current thread

		unsigned int idx=c*rows1+r;	//column major index, using row and column value
		
		float val=0;//register to multiplication result

		for(int m=0; m<1+((rows2-1)/TILE_WIDTH);m++)	//going over all tiles one by one, with each m
		{

			int var1=m*TILE_WIDTH+tx ;		//x thread value for current tile
			int var2=m*TILE_WIDTH+ty ;		//y thread value for current tile
			
			if (r < rows1 && var1 < rows2)	//if the value is associated to a valid matrix coordinate then store it to shared, else store zero
				S1[ty][tx]=array1[r + var1*rows1];	//storing a "valid" value from array to shared memory
			else
					S1[ty][tx]=0;					//storing zero, since there is no valid value
       		__syncthreads();						//syncing all threads once shared memory S1 is stored
			
       		if(c<cols2 && var2< rows2)
      			S2[ty][tx]=array2[var2+rows2*c];
      		else 
      			{
      			S2[ty][tx]=0;
      			// printf("S2 is zero\n");
      		}
			__syncthreads();
			

			for(int i=0; i<TILE_WIDTH;i++)
				val+=S1[ty][i]*S2[i][tx];
			__syncthreads();

		}
		
		if(r < rows1 && c< cols2)	
			array3[idx]=val;
			
	}

	int main(int argc, char* argv[])
	{
		if(argc != 4) //there should be four arguments
		return 1; //exit and return an error

		time_t reading_start=time(NULL);

		ifstream infile_A, infile_B;	//reading the input matrices
		
		
		//READING matrix A
		infile_A.open(argv[1],ios::binary|ios::in|ios::ate);
		
		//getting end and beginning of the file
		infile_A.seekg(0,ios::end);
		infile_A.seekg(0,ios::beg);
		
		//memory allocation
		matrix M_A;	
		infile_A.read(reinterpret_cast<char*>(&M_A),2*sizeof(unsigned int));
		

		float* array_A=(float*)malloc(M_A.rows*M_A.cols*sizeof(float));	//column major
		infile_A.read(reinterpret_cast<char*>(array_A),M_A.rows*M_A.cols*sizeof(float));
		
		infile_A.close();

		//READING matrix B
		infile_B.open(argv[2],ios::binary|ios::in|ios::ate);
		
		//getting end and beginning of the file
		infile_B.seekg(0,ios::end);
		infile_B.seekg(0,ios::beg);
		
		//memory allocation
		matrix M_B;
		infile_B.read(reinterpret_cast<char*>(&M_B),2*sizeof(unsigned int));

		float* array_B=(float*)malloc(M_B.rows*M_B.cols*sizeof(float));	//column major

		infile_B.read(reinterpret_cast<char*>(array_B),M_B.rows*M_B.cols*sizeof(float));
		
		infile_B.close();

		if(M_A.cols!=M_B.rows)
		{
			cout<<"Illegal matrix sizes: "<<M_A.cols<<" != "<<M_B.rows<<endl;
			return 1;
		}

		float* array_C=(float*)malloc(M_A.rows*M_B.cols*sizeof(float));//gpu result
		
		float* array_D=(float*)malloc(M_A.rows*M_B.cols*sizeof(float));//cublas result
		
		time_t reading_end = time(NULL);


		//GPU DEVICE PROPERTIES
		int nDevices;
		HANDLE_ERROR(hipGetDeviceCount(&nDevices));

		hipDeviceProp_t prop;
	   	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));	//using GPU0

	   	//BLOCK AND GRID SIZE
	   	float thread_block=sqrt(prop.maxThreadsPerBlock);
		dim3 DimGrid(ceil(M_B.cols/thread_block),ceil(M_A.rows/thread_block),1); //image saved as a 2D grid
		dim3 DimBlock(thread_block,thread_block,1);

		size_t Sbytes = 2* DimBlock.x * DimBlock.y ;
		

		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, 0);

		if(props.sharedMemPerBlock < Sbytes){
			std::cout<<"ERROR: insufficient shared memory"<<std::endl;
			exit(1);
		}

		//GPU MEMORY ALLOCATION
		float *array_A_gpu, *array_B_gpu, *array_C_gpu, *array_D_gpu;
	   
		HANDLE_ERROR(hipMalloc(&array_A_gpu,M_A.rows*M_A.cols*sizeof(float))); //allocate space to store convolution result

		HANDLE_ERROR(hipMalloc(&array_B_gpu,M_B.rows*M_B.cols*sizeof(float))); //allocate space to store convolution temporary

		HANDLE_ERROR(hipMalloc(&array_C_gpu,M_A.rows*M_B.cols*sizeof(float))); //allocate space to copy image to GPU memory

		HANDLE_ERROR(hipMalloc(&array_D_gpu,M_A.rows*M_B.cols*sizeof(float))); //allocate space to copy image to GPU memory


		//COPY TO GPU MEMORY
		HANDLE_ERROR(hipMemcpy(array_A_gpu, array_A, M_A.rows*M_A.cols*sizeof(float), hipMemcpyHostToDevice));//copy input image from global to gpu

		HANDLE_ERROR(hipMemcpy(array_B_gpu, array_B, M_B.rows*M_B.cols*sizeof(float), hipMemcpyHostToDevice));//copy the kernel0 host to device

		HANDLE_ERROR(hipMemcpy(array_C_gpu, array_C, M_A.rows*M_B.cols*sizeof(float), hipMemcpyHostToDevice));//copy kernel1 host to device

		HANDLE_ERROR(hipMemcpy(array_D_gpu, array_D, M_A.rows*M_B.cols*sizeof(float), hipMemcpyHostToDevice));//copy kernel1 host to device

		time_t memory_transfers=time(NULL);

		//time measurement
		hipEvent_t start1, stop1;
	 	
	 	hipEventCreate(&start1);
		hipEventCreate(&stop1);
		
		//MATRIX MULTIPLICATION
		hipEventRecord(start1);
		matrix_mult<<<DimGrid, DimBlock, Sbytes>>>(array_A_gpu,M_A.rows,M_A.cols,array_B_gpu,M_B.rows,M_B.cols,array_C_gpu);
		hipEventRecord(stop1);

		time_t mult_end = time(NULL);

		hipEventSynchronize(stop1);
		float milliseconds1 = 0, milliseconds2 = 0;
		
		hipEventElapsedTime(&milliseconds1, start1, stop1);
		cout<<"time taken by GPU = "<<milliseconds1<<" ms"<<endl;

		//copy to CPU MEMORY
		HANDLE_ERROR(hipMemcpy(array_C, array_C_gpu, M_A.rows*M_B.cols*sizeof(float), hipMemcpyDeviceToHost));//copy kernel1 host to device

		//Creating handle for CUBLAS
		hipblasHandle_t handle;
		hipblasCreate(&handle);	

		float alpha = 1.0;
		float beta = 0.0;
	    
	    hipEvent_t start2, stop2;
	 	
	 	hipEventCreate(&start2);
		hipEventCreate(&stop2);

		hipEventRecord(start2);
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M_A.rows, M_B.cols, M_A.cols, &alpha, array_A_gpu, M_A.rows, array_B_gpu, M_B.rows, &beta, array_D_gpu, M_A.rows);
		hipEventRecord(stop2);

		hipEventSynchronize(stop2);

		hipEventElapsedTime(&milliseconds2, start2, stop2);
		cout<<"time taken by CUBLAS= "<<milliseconds2<<" ms"<<endl;
		
		//copy to CPU MEMORY
	    
	    HANDLE_ERROR(hipMemcpy(array_D, array_D_gpu, M_A.rows*M_B.cols*sizeof(float), hipMemcpyDeviceToHost));//copy kernel1 host to device

		float mse=0; //mean squared error;

		for(int i=0; i<M_A.rows*M_B.cols;i++)
			mse=mse+(array_C[i]-array_D[i])*(array_C[i]-array_D[i]);
			
		cout<<endl<<"Mean square error = "<<mse<<endl;

		//SAVING THE OUTPUT MATRIX
		ofstream ofile(argv[3], ios::binary);

		ofile.write((char*) &M_A.rows, sizeof(unsigned int));
		ofile.write((char*) &M_B.cols, sizeof(unsigned int));	
		ofile.write((char*) array_C , M_A.rows*M_B.cols*sizeof(float))	;

		time_t saved = time(NULL);

		// cout<<"Matrix reading     :"<<double(reading_end - reading_start)<<" secs"<<endl;
		// cout<<"Memory Transfers   :"<<double(memory_transfers - reading_end)<<" secs"<<endl;
		// cout<<"Multiplication done:"<<double(mult_end - memory_transfers)<<" secs"<<endl;
		// cout<<"Matrix saving      :"<<double(saved - mult_end)<<" secs"<<endl;

		return 0;
	}