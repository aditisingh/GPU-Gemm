#include <fstream>
#include <iostream>
#include <stdio.h>
#include <string>
#include <sstream>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <ctime>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

using namespace std;

struct matrix{
	unsigned int rows;
	unsigned int cols;
};

static void HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		cout<<hipGetErrorString(err)<<" in "<< file <<" at line "<< line<<endl;
	}
}


#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__ void matrix_mult(float* array1, unsigned int rows1, unsigned int cols1, float* array2, unsigned int rows2, unsigned int cols2, float* array3)
{
	 extern __shared__ float S[];			//defined a shared memory pointer
	
	size_t c=blockIdx.x*blockDim.x + threadIdx.x;
	size_t r=blockIdx.y*blockDim.y + threadIdx.y;

	if(r>=rows1 || c>=cols2) return;

	size_t idx=c*cols2+r;	//going columnwise
	size_t C1=rows2*c;
	
	float val=0;

	for(int i=0; i<rows1*cols1;i++)
		S[i]=array1[i];

	for(int i=0; i<rows2*cols2;i++)
		S[i+rows1*cols1]=array2[i];

	#pragma unroll 8
	for(int k=0;k<rows2;k++)
	{
		val+=S[rows1*k+r]*S[rows1*cols1+C1+k];
	}
	array3[idx]=val;

}

int main(int argc, char* argv[])
{
	if(argc != 4) //there should be four arguments
	return 1; //exit and return an error

	time_t reading_start=time(NULL);

	ifstream infile_A, infile_B;	//reading the input matrices
	
	
	//READING matrix A
	infile_A.open(argv[1],ios::binary|ios::in|ios::ate);
	
	//getting end and beginning of the file
	infile_A.seekg(0,ios::end);
	infile_A.seekg(0,ios::beg);
	
	//memory allocation
	matrix M_A;	
	infile_A.read(reinterpret_cast<char*>(&M_A),2*sizeof(unsigned int));
	//cout<<M_A.rows<<M_A.cols;
	
	//matrix M_A,M_B;

	float* array_A=(float*)malloc(M_A.rows*M_A.cols*sizeof(float));	//column major
	infile_A.read(reinterpret_cast<char*>(array_A),M_A.rows*M_A.cols*sizeof(float));
	
	infile_A.close();

	//READING matrix B
	infile_B.open(argv[1],ios::binary|ios::in|ios::ate);
	
	//getting end and beginning of the file
	infile_B.seekg(0,ios::end);
	infile_B.seekg(0,ios::beg);
	
	//memory allocation
	matrix M_B;
	infile_B.read(reinterpret_cast<char*>(&M_B),2*sizeof(unsigned int));

	float* array_B=(float*)malloc(M_B.rows*M_B.cols*sizeof(float));	//column major

/*	array_A[0]=1, array_A[3]=2, array_A[6]=1;
	array_A[1]=2, array_A[4]=3, array_A[7]=4;
	array_A[2]=1, array_A[5]=-1, array_A[8]=0;

	array_B[0]=0, array_B[3]=1, array_B[6]=0;
	array_B[1]=1, array_B[4]=2, array_B[7]=3;
	array_B[2]=-1, array_B[5]=2, array_B[8]=-1;

	*/

	infile_B.read(reinterpret_cast<char*>(array_B),M_B.rows*M_B.cols*sizeof(float));
	
	infile_B.close();

	if(M_A.cols!=M_B.rows)
	{
		cout<<"Illegal matrix sizes: "<<M_A.cols<<" != "<<M_B.rows<<endl;
		return 1;
	}

	float* array_C=(float*)malloc(M_A.rows*M_B.cols*sizeof(float));//gpu result
	
	float* array_D=(float*)malloc(M_A.rows*M_B.cols*sizeof(float));//cublas result
	
	time_t reading_end = time(NULL);


	//GPU DEVICE PROPERTIES
	int nDevices;
	HANDLE_ERROR(hipGetDeviceCount(&nDevices));

	hipDeviceProp_t prop;
   	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));	//using GPU0

   	//BLOCK AND GRID SIZE
    float thread_block=sqrt(prop.maxThreadsPerBlock);
	dim3 DimGrid(ceil(M_B.cols/thread_block),ceil(M_A.rows/thread_block),1); //image saved as a 2D grid
	dim3 DimBlock(thread_block,thread_block,1);

	size_t Sbytes = DimBlock.x * DimBlock.y * sizeof(float) * 2;

	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, 0);
	if(props.sharedMemPerBlock < Sbytes){
		std::cout<<"ERROR: insufficient shared memory"<<std::endl;
		exit(1);
	}

	//GPU MEMORY ALLOCATION
	float *array_A_gpu, *array_B_gpu, *array_C_gpu, *array_D_gpu;
   
	HANDLE_ERROR(hipMalloc(&array_A_gpu,M_A.rows*M_A.cols*sizeof(float))); //allocate space to store convolution result

	HANDLE_ERROR(hipMalloc(&array_B_gpu,M_B.rows*M_B.cols*sizeof(float))); //allocate space to store convolution temporary

	HANDLE_ERROR(hipMalloc(&array_C_gpu,M_A.rows*M_B.cols*sizeof(float))); //allocate space to copy image to GPU memory

	HANDLE_ERROR(hipMalloc(&array_D_gpu,M_A.rows*M_B.cols*sizeof(float))); //allocate space to copy image to GPU memory


	//COPY TO GPU MEMORY
	HANDLE_ERROR(hipMemcpy(array_A_gpu, array_A, M_A.rows*M_A.cols*sizeof(float), hipMemcpyHostToDevice));//copy input image from global to gpu

	HANDLE_ERROR(hipMemcpy(array_B_gpu, array_B, M_B.rows*M_B.cols*sizeof(float), hipMemcpyHostToDevice));//copy the kernel0 host to device

	HANDLE_ERROR(hipMemcpy(array_C_gpu, array_C, M_A.rows*M_B.cols*sizeof(float), hipMemcpyHostToDevice));//copy kernel1 host to device

	HANDLE_ERROR(hipMemcpy(array_D_gpu, array_D, M_A.rows*M_B.cols*sizeof(float), hipMemcpyHostToDevice));//copy kernel1 host to device

	time_t memory_transfers=time(NULL);

	//time measurement
	hipEvent_t start1, stop1;
 	
 	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	
	//MATRIX MULTIPLICATION
	hipEventRecord(start1);
	matrix_mult<<<DimGrid, DimBlock, Sbytes>>>(array_A_gpu,M_A.rows,M_A.cols,array_B_gpu,M_B.rows,M_B.cols,array_C_gpu);
	hipEventRecord(stop1);

	time_t mult_end = time(NULL);

	hipEventSynchronize(stop1);
	float milliseconds1 = 0, milliseconds2 = 0;
	
	hipEventElapsedTime(&milliseconds1, start1, stop1);
	cout<<"time taken by GPU = "<<milliseconds1<<" ms"<<endl;

	//copy to CPU MEMORY
	HANDLE_ERROR(hipMemcpy(array_C, array_C_gpu, M_A.rows*M_B.cols*sizeof(float), hipMemcpyDeviceToHost));//copy kernel1 host to device

	//Creating handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);	

	float alpha = 1.0;
	float beta = 0.0;
    
    hipEvent_t start2, stop2;
 	
 	hipEventCreate(&start2);
	hipEventCreate(&stop2);

	hipEventRecord(start2);
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M_A.rows, M_B.cols, M_A.cols, &alpha, array_A_gpu, M_A.rows, array_B_gpu, M_B.rows, &beta, array_D_gpu, M_A.rows);
	hipEventRecord(stop2);

	hipEventSynchronize(stop2);

	hipEventElapsedTime(&milliseconds2, start2, stop2);
	cout<<"time taken by CUBLAS= "<<milliseconds2<<" ms"<<endl;
	
	//copy to CPU MEMORY
    
    HANDLE_ERROR(hipMemcpy(array_D, array_D_gpu, M_A.rows*M_B.cols*sizeof(float), hipMemcpyDeviceToHost));//copy kernel1 host to device

	float mse=0; //mean squared error

	for(int i=0; i<M_A.rows*M_B.cols;i++)
		{
		mse=mse+(array_C[i]-array_D[i])*(array_C[i]-array_D[i]);
		//float diff=array_C[i]-array_D[i];
		//cout<<diff<<" ";//
		// cout<<array_C[i]<<" "<<" "<<array_D[i]<<endl;
		}

	cout<<endl<<"Mean square error = "<<mse<<endl;

	//SAVING THE OUTPUT MATRIX
	ofstream ofile(argv[3], ios::binary);

	ofile.write((char*) &M_A.rows, sizeof(unsigned int));
	ofile.write((char*) &M_B.cols, sizeof(unsigned int));	
	ofile.write((char*) array_C , M_A.rows*M_B.cols*sizeof(float))	;

	time_t saved = time(NULL);

	//cout<<"Matrix reading     :"<<double(reading_end - reading_start)<<" secs"<<endl;
	//cout<<"Memory Transfers   :"<<double(memory_transfers - reading_end)<<" secs"<<endl;
	//cout<<"Multiplication done:"<<double(mult_end - memory_transfers)<<" secs"<<endl;
	//cout<<"Matrix saving      :"<<double(saved - mult_end)<<" secs"<<endl;

	return 0;
}
