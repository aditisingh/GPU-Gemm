	#include <fstream>
	#include <iostream>
	#include <stdio.h>
	#include <string>
	#include <sstream>
	#include <stdlib.h>
	#include <math.h>
	#include <time.h>
	#include <ctime>
	#include <vector>
	#include <hip/hip_runtime.h>
	#include <hip/hip_runtime_api.h>
	#include <hipblas.h>

	using namespace std;

	struct matrix{
		unsigned int rows;
		unsigned int cols;
	};

	static void HandleError( hipError_t err, const char *file, int line ) {
		if (err != hipSuccess) {
			cout<<hipGetErrorString(err)<<" in "<< file <<" at line "<< line<<endl;
		}
	}


	#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


	__global__ void matrix_mult(float* array1, unsigned int rows1, unsigned int cols1, float* array2, unsigned int rows2, unsigned int cols2, float* array3)
	{	
		//each tile should fit the shared memory

		const unsigned int TILE_WIDTH = 32;
		const unsigned int TILE_HEIGHT= 32;//32

		__shared__ float S1[TILE_WIDTH][TILE_HEIGHT];
		__shared__ float S2[TILE_HEIGHT][TILE_WIDTH];

		size_t tx=threadIdx.x;
		size_t ty=threadIdx.y;

		const unsigned int bx=blockIdx.x*blockDim.x;   //upper left corner x index
		const unsigned int by=blockIdx.y*blockDim.y;	//upper left corner y index

		size_t c=bx + tx;	//x-index of current thread
		size_t r=by + ty;	//y-index of current thread

		size_t idx=c*cols2+r;

		float val=0;

		for(int m=0; m<cols2/TILE_WIDTH;m++)
		{
			S1[ty][tx]=array1[r + (m*TILE_WIDTH+tx)*rows1];
			S2[ty][tx]=array2[(m*TILE_WIDTH+ty)+cols2*c];
			__syncthreads();

			for(int i=0; i<TILE_WIDTH;i++)
				val+=S1[ty][i]*S2[i][tx];
			__syncthreads();

		}
		
		//for(int i=0;i<TILE_WIDTH;i++)
		//{	
		//	for(int j=0;j<TILE_HEIGHT;j++)
		//	{	
				//printf("S1 %d, S2 %d \t",S1[i][j],S2[i][j]);
		//	}
		//}
	
		//printf("\n");
		//printf("%d,\t",S1[ty][tx]);
		//printf("%d, \n",S2[ty][tx]);
		
		array3[idx]=val;

	}

	int main(int argc, char* argv[])
	{
		if(argc != 4) //there should be four arguments
		return 1; //exit and return an error

		time_t reading_start=time(NULL);

		ifstream infile_A, infile_B;	//reading the input matrices
		
		
		//READING matrix A
		infile_A.open(argv[1],ios::binary|ios::in|ios::ate);
		
		//getting end and beginning of the file
		infile_A.seekg(0,ios::end);
		infile_A.seekg(0,ios::beg);
		
		//memory allocation
		matrix M_A;	
		infile_A.read(reinterpret_cast<char*>(&M_A),2*sizeof(unsigned int));
		//cout<<M_A.rows<<M_A.cols;
		
		//matrix M_A,M_B;
		// M_A.rows=4, M_A.cols=6;

		float* array_A=(float*)malloc(M_A.rows*M_A.cols*sizeof(float));	//column major
		infile_A.read(reinterpret_cast<char*>(array_A),M_A.rows*M_A.cols*sizeof(float));
		
		infile_A.close();

		//READING matrix B
		infile_B.open(argv[2],ios::binary|ios::in|ios::ate);
		
		//getting end and beginning of the file
		infile_B.seekg(0,ios::end);
		infile_B.seekg(0,ios::beg);
		
		//memory allocation
		matrix M_B;
		infile_B.read(reinterpret_cast<char*>(&M_B),2*sizeof(unsigned int));

		//M_B.rows=6, M_B.cols=4;

		float* array_B=(float*)malloc(M_B.rows*M_B.cols*sizeof(float));	//column major

		// array_A[0]=1, array_A[3]=2, array_A[6]=1;
		// array_A[1]=2, array_A[4]=3, array_A[7]=4;
	 //    array_A[2]=1, array_A[5]=-1, array_A[8]=0;

		// array_B[0]=0, array_B[3]=1, array_B[6]=0;
		// array_B[1]=1, array_B[4]=2, array_B[7]=3;
		// array_B[2]=-1, array_B[5]=2, array_B[8]=-1;

		infile_B.read(reinterpret_cast<char*>(array_B),M_B.rows*M_B.cols*sizeof(float));
		
		infile_B.close();

		// array_B[0]=0, array_B[3]=1, array_B[6]=0;
		// array_B[1]=1, array_B[4]=2, array_B[7]=3;
		// array_B[2]=-1, array_B[5]=2, array_B[8]=-1;
	
	//array_A[0]=0, array_A[1]=2, array_A[2]=3, array_A[3]=0, array_A[4]=3, array_A[5]= 5;
       //  	array_A[6]=1, array_A[7]=0, array_A[8]=4, array_A[9]=6, array_A[10]=7,array_A[11]=7;
	//	array_A[12]=0, array_A[13]=2, array_A[14]=2, array_A[15]=3, array_A[16]=7,array_A[17]=0;
  // 		array_A[18]=3, array_A[19]=0, array_A[20]=2, array_A[21]=1, array_A[22]=1,array_A[23]=5; 
//

	//	array_B[0]=1, array_B[1]=4, array_B[2]=5, array_B[3]=1; 	
	  //      array_B[4]=8, array_B[5]=3, array_B[6]=2, array_B[7]=5;
     		//array_B[8]=6, array_B[9]=2, array_B[10]=0,array_B[11]=1;
	//	array_B[12]=4,array_B[13]=2,array_B[14]=8,array_B[15]=3;
	//	array_B[16]=4,array_B[17]=2,array_B[18]=1,array_B[19]=1;
	//	array_B[20]=8,array_B[21]=4,array_B[22]=1,array_B[23]=6;
    
		if(M_A.cols!=M_B.rows)
		{
			cout<<"Illegal matrix sizes: "<<M_A.cols<<" != "<<M_B.rows<<endl;
			return 1;
		}

		float* array_C=(float*)malloc(M_A.rows*M_B.cols*sizeof(float));//gpu result

		//initialise it to zero
		for(int i=0; i<M_B.cols*M_A.rows;i++)
			array_C[i]=0;
		
		float* array_D=(float*)malloc(M_A.rows*M_B.cols*sizeof(float));//cublas result
		
		time_t reading_end = time(NULL);


		//GPU DEVICE PROPERTIES
		int nDevices;
		HANDLE_ERROR(hipGetDeviceCount(&nDevices));

		hipDeviceProp_t prop;
	   	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));	//using GPU0

	   	//BLOCK AND GRID SIZE
	    float thread_block=sqrt(prop.maxThreadsPerBlock);
		dim3 DimGrid(ceil(M_B.cols/thread_block),ceil(M_A.rows/thread_block),1); //image saved as a 2D grid
		dim3 DimBlock(thread_block,thread_block,1);

		size_t Sbytes = 2* DimBlock.x * DimBlock.y ;
		

		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, 0);
		cout<<Sbytes<<" "<<props.sharedMemPerBlock<<endl;
		if(props.sharedMemPerBlock < Sbytes){
			std::cout<<"ERROR: insufficient shared memory"<<std::endl;
			exit(1);
		}

		//GPU MEMORY ALLOCATION
		float *array_A_gpu, *array_B_gpu, *array_C_gpu, *array_D_gpu;
	   
		HANDLE_ERROR(hipMalloc(&array_A_gpu,M_A.rows*M_A.cols*sizeof(float))); //allocate space to store convolution result

		HANDLE_ERROR(hipMalloc(&array_B_gpu,M_B.rows*M_B.cols*sizeof(float))); //allocate space to store convolution temporary

		HANDLE_ERROR(hipMalloc(&array_C_gpu,M_A.rows*M_B.cols*sizeof(float))); //allocate space to copy image to GPU memory

		HANDLE_ERROR(hipMalloc(&array_D_gpu,M_A.rows*M_B.cols*sizeof(float))); //allocate space to copy image to GPU memory


		//COPY TO GPU MEMORY
		HANDLE_ERROR(hipMemcpy(array_A_gpu, array_A, M_A.rows*M_A.cols*sizeof(float), hipMemcpyHostToDevice));//copy input image from global to gpu

		HANDLE_ERROR(hipMemcpy(array_B_gpu, array_B, M_B.rows*M_B.cols*sizeof(float), hipMemcpyHostToDevice));//copy the kernel0 host to device

		HANDLE_ERROR(hipMemcpy(array_C_gpu, array_C, M_A.rows*M_B.cols*sizeof(float), hipMemcpyHostToDevice));//copy kernel1 host to device

		HANDLE_ERROR(hipMemcpy(array_D_gpu, array_D, M_A.rows*M_B.cols*sizeof(float), hipMemcpyHostToDevice));//copy kernel1 host to device

		time_t memory_transfers=time(NULL);

		//time measurement
		hipEvent_t start1, stop1;
	 	
	 	hipEventCreate(&start1);
		hipEventCreate(&stop1);
		
		//MATRIX MULTIPLICATION
		hipEventRecord(start1);
		matrix_mult<<<DimGrid, DimBlock, Sbytes>>>(array_A_gpu,M_A.rows,M_A.cols,array_B_gpu,M_B.rows,M_B.cols,array_C_gpu);
		hipEventRecord(stop1);

		time_t mult_end = time(NULL);

		hipEventSynchronize(stop1);
		float milliseconds1 = 0, milliseconds2 = 0;
		
		hipEventElapsedTime(&milliseconds1, start1, stop1);
		cout<<"time taken by GPU = "<<milliseconds1<<" ms"<<endl;

		//copy to CPU MEMORY
		HANDLE_ERROR(hipMemcpy(array_C, array_C_gpu, M_A.rows*M_B.cols*sizeof(float), hipMemcpyDeviceToHost));//copy kernel1 host to device

		//Creating handle for CUBLAS
		hipblasHandle_t handle;
		hipblasCreate(&handle);	

		float alpha = 1.0;
		float beta = 0.0;
	    
	    hipEvent_t start2, stop2;
	 	
	 	hipEventCreate(&start2);
		hipEventCreate(&stop2);

		hipEventRecord(start2);
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M_A.rows, M_B.cols, M_A.cols, &alpha, array_A_gpu, M_A.rows, array_B_gpu, M_B.rows, &beta, array_D_gpu, M_A.rows);
		hipEventRecord(stop2);

		hipEventSynchronize(stop2);

		hipEventElapsedTime(&milliseconds2, start2, stop2);
		cout<<"time taken by CUBLAS= "<<milliseconds2<<" ms"<<endl;
		
		//copy to CPU MEMORY
	    
	    HANDLE_ERROR(hipMemcpy(array_D, array_D_gpu, M_A.rows*M_B.cols*sizeof(float), hipMemcpyDeviceToHost));//copy kernel1 host to device

		float mse=0; //mean squared error

		for(int i=0; i<M_A.rows*M_B.cols;i++)
			{
			mse=mse+(array_C[i]-array_D[i])*(array_C[i]-array_D[i]);
			//float diff=array_C[i]-array_D[i];
			//cout<<diff<<" ";//
			 //cout<<array_A[i]<<" "<<array_B[i]
			//cout<<i<<" "<<array_C[i]<<" "<<" "<<array_D[i]<<endl;
			}

		cout<<endl<<"Mean square error = "<<mse<<endl;

		//SAVING THE OUTPUT MATRIX
		ofstream ofile(argv[3], ios::binary);

		ofile.write((char*) &M_A.rows, sizeof(unsigned int));
		ofile.write((char*) &M_B.cols, sizeof(unsigned int));	
		ofile.write((char*) array_C , M_A.rows*M_B.cols*sizeof(float))	;

		time_t saved = time(NULL);

		//cout<<"Matrix reading     :"<<double(reading_end - reading_start)<<" secs"<<endl;
		//cout<<"Memory Transfers   :"<<double(memory_transfers - reading_end)<<" secs"<<endl;
		//cout<<"Multiplication done:"<<double(mult_end - memory_transfers)<<" secs"<<endl;
		//cout<<"Matrix saving      :"<<double(saved - mult_end)<<" secs"<<endl;

		return 0;
	}
